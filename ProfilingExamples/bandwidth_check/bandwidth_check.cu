#include <cstdio>
#include <cstdlib>
#include <getopt.h>
#include <hip/hip_runtime.h>
#include <vector>

// Default values
constexpr int DEFAULT_NUM_COPY_ITERATIONS = 1000;
constexpr int DEFAULT_NUM_STREAMS = 4;
constexpr int DEFAULT_WARMUP_ITERATIONS = 10;
constexpr size_t DEFAULT_DATA_SIZE = 1 << 24; // 16M floats

void checkBandwidth(size_t dataSize, int numCopyIterations, int numStreams, int warmupIterations) {
    // Allocate pinned host memory.
    float *hData;
    ::hipHostAlloc(&hData, dataSize * sizeof(float), hipHostMallocDefault);

    // Populate the host array with values.
    for (size_t i = 0; i < dataSize; ++i) {
        hData[i] = static_cast<float>(i);
    }

    // Allocate device memory.
    float *dData;
    ::hipMalloc(&dData, dataSize * sizeof(float));

    // Create CUDA events for timing purposes.
    hipEvent_t start, stop;
    hipEventCreatete(&start);
    hipEventCreatete(&stop);

    // Create multiple CUDA streams.
    std::vector<hipStream_t> streams(numStreams);
    for (int i = 0; i < numStreams; ++i) {
        ::hipStreamCreate(&streams[i]);
    }

    // Perform warm-up iterations to stabilize performance.
    for (int i = 0; i < warmupIterations; ++i) {
        int streamIndex = i % numStreams;
        ::hipMemcpyAsync(dData, hData, dataSize * sizeof(float), hipMemcpyHostToDevice, streams[streamIndex]);
    }
    for (int i = 0; i < numStreams; ++i) {
        ::hipStreamSynchronize(streams[i]);
    }

    // Record the start event.
    ::hipEventRecord(start);

    // Perform data transfers using multiple streams.
    for (int i = 0; i < numCopyIterations; ++i) {
        int streamIndex = i % numStreams;
        ::hipMemcpyAsync(dData, hData, dataSize * sizeof(float), hipMemcpyHostToDevice, streams[streamIndex]);
    }

    // Sync all streams to ensure copy complete.
    for (int i = 0; i < numStreams; ++i) {
        ::hipStreamSynchronize(streams[i]);
    }

    // Record the stop event.
    ::hipEventRecord(stop);

    // Wait for the stop event to complete.
    ::hipEventSynchronize(stop);

    // Calculate the elapsed time.
    float ms = 0;
    ::hipEventElapsedTime(&ms, start, stop);

    // Check if ms is zero to avoid division by zero.
    if (ms > 0) {
        // Convert to GB/s.
        const float bandwidth = ((float(dataSize) * sizeof(float) * float(numCopyIterations)) / (ms * 1e6));
        printf("Bandwidth: %f GB/s\n", bandwidth);
    } else {
        printf("Bandwidth calculation error: elapsed time is zero.\n");
    }

    // Clean up.
    ::hipFree(dData);
    ::hipHostFree(hData);
    for (int i = 0; i < numStreams; ++i) {
        ::hipStreamDestroy(streams[i]);
    }
    ::hipEventDestroy(start);
    ::hipEventDestroy(stop);
}

int main(int argc, char* argv[]) {
    // Command-line options
    int numCopyIterations = DEFAULT_NUM_COPY_ITERATIONS;
    int numStreams = DEFAULT_NUM_STREAMS;
    int warmupIterations = DEFAULT_WARMUP_ITERATIONS;
    size_t dataSize = DEFAULT_DATA_SIZE;

    // Parse command-line arguments
    int option;
    while ((option = getopt(argc, argv, "i:s:w:d:")) != -1) {
        switch (option) {
            case 'i':
                numCopyIterations = std::atoi(optarg);
                break;
            case 's':
                numStreams = std::atoi(optarg);
                break;
            case 'w':
                warmupIterations = std::atoi(optarg);
                break;
            case 'd':
                dataSize = std::atol(optarg);
                break;
            default:
                fprintf(stderr, "Usage: %s [-i iterations] [-s streams] [-w warmup] [-d dataSize]\n", argv[0]);
                fprintf(stderr, "  -i iterations : Number of copy iterations (default: %d)\n", DEFAULT_NUM_COPY_ITERATIONS);
                fprintf(stderr, "  -s streams    : Number of streams (default: %d)\n", DEFAULT_NUM_STREAMS);
                fprintf(stderr, "  -w warmup     : Number of warm-up iterations (default: %d)\n", DEFAULT_WARMUP_ITERATIONS);
                fprintf(stderr, "  -d dataSize   : Data size in number of floats (default: %zu)\n", DEFAULT_DATA_SIZE);
                fprintf(stderr, "Examples:\n");
                fprintf(stderr, "  %s -i 2000 -s 8 -w 20 -d 33554432\n", argv[0]);
                fprintf(stderr, "  %s -d 16777216\n", argv[0]);
                exit(EXIT_FAILURE);
        }
    }

    checkBandwidth(dataSize, numCopyIterations, numStreams, warmupIterations);
    return 0;
}
